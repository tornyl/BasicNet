
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


void get_max_blocks_per_grid(int device_id = 0) {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);

    int max_threads_per_block = props.maxThreadsPerBlock;
    int max_threads_dim = props.maxThreadsDim[0]; // Assuming a 1D grid
    int max_blocks_dim = max_threads_dim / max_threads_per_block;
    int max_blocks_per_grid = max_blocks_dim;

    std::cout << "global memory " << props.totalGlobalMem << std::endl;
    std::cout << "grid size " << props.maxGridSize[0] << std::endl;



    std::cout << "Maximum number of threads per block: " << max_threads_per_block << std::endl;
    std::cout << "Maximum number of threads per dim: " << max_threads_dim << std::endl;
    std::cout << "Maximum number of blocks per dim: " << max_blocks_dim << std::endl;
    std::cout << "Maximum number of blocks per grid: " << max_blocks_per_grid << std::endl;
}




void gpu_memory_info(){
    size_t free_bytes, total_bytes;
    hipError_t cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);
    if (cuda_status != hipSuccess) {
        std::cerr << "Error getting CUDA memory info" << std::endl;
    }

    std::cout << "Total GPU memory: " << total_bytes << " bytes" << std::endl;
    std::cout << "Free GPU memory: " << free_bytes << " bytes" << std::endl;

}

int main(){

    cout<<"Hello world"<<endl;
    cuda_hello<<<1,1>>>();

    gpu_memory_info();
    get_max_blocks_per_grid();

    return 0;

}



